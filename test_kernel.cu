#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <nvToolsExt.h>

#include <hip/hip_runtime_api.h>

#include <stdio.h>

#include <pthread.h>

//#include "utils.h"



#define BLOCKS 80

#define THREADS 512

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }   \
} 


__global__ void bar() {

	__shared__ int a[THREADS];

	int tid = threadIdx.x;

	for (int i = 0; i < 1000000; i++) {

		a[tid] += tid + i;

	}

//	if (tid==10) printf("%d\n", a[tid]);

}



int foo() {

	int sum = 0;

	for(int i = 0; i <  1000000; i++) {

		sum += i;

	}

	return sum;

}



int main(void) {

	hipProfilerStart();


	nvtxNameOsThread(pthread_self(), "MAIN");



	nvtxRangePush("Calling foo");

	printf("%d\n", foo());

	nvtxRangePop();



	nvtxRangePush("Calling bar1");

	bar<<<BLOCKS, THREADS>>>();

	nvtxRangePop();

        cudaCheckError();


	nvtxRangePush("Calling bar2");

	bar<<<BLOCKS, THREADS>>>();

	nvtxRangePop();

        cudaCheckError();



	nvtxRangePush("Calling bar3");

	bar<<<BLOCKS, THREADS>>>();

	nvtxRangePop();

        cudaCheckError();



	hipDeviceSynchronize();



	hipProfilerStop();

	return 0;

}
